#include "hip/hip_runtime.h"
/* vim: set sw=4 sts=4 et nofoldenable : */

/*
 * Copyright (c) 2008 Dirk Ribbrock <dirk.ribbrock@uni-dortmund.de>
 *
 * This file is part of the HONEI C++ library. HONEI is free software;
 * you can redistribute it and/or modify it under the terms of the GNU General
 * Public License version 2, as published by the Free Software Foundation.
 *
 * HONEI is distributed in the hope that it will be useful, but WITHOUT ANY
 * WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more
 * details.
 *
 * You should have received a copy of the GNU General Public License along with
 * this program; if not, write to the Free Software Foundation, Inc., 59 Temple
 * Place, Suite 330, Boston, MA  02111-1307  USA
 */

#include <honei/backends/cuda/cuda_util.hh>

namespace honei
{
    namespace cuda
    {
        __global__ void difference_gpu(float * x, float * y, unsigned long size)
        {
            unsigned long idx = (blockDim.y * blockIdx.y * gridDim.x * blockDim.x) + (blockDim.x * blockIdx.x) + threadIdx.x;
            if (idx < size)
            {
                x[idx] = x[idx] - y[idx];
            }
        }

#ifdef HONEI_CUDA_DOUBLE
        __global__ void difference_gpu(double * x, double * y, unsigned long size)
        {
            unsigned long idx = (blockDim.y * blockIdx.y * gridDim.x * blockDim.x) + (blockDim.x * blockIdx.x) + threadIdx.x;
            if (idx < size)
            {
                x[idx] = x[idx] - y[idx];
            }
        }
#endif

        __global__ void difference_gpu(float * r, float * x, float * y, unsigned long size)
        {
            unsigned long idx = (blockDim.y * blockIdx.y * gridDim.x * blockDim.x) + (blockDim.x * blockIdx.x) + threadIdx.x;
            if (idx < size)
            {
                r[idx] = x[idx] - y[idx];
            }
        }

#ifdef HONEI_CUDA_DOUBLE
        __global__ void difference_gpu(double * r, double * x, double * y, unsigned long size)
        {
            unsigned long idx = (blockDim.y * blockIdx.y * gridDim.x * blockDim.x) + (blockDim.x * blockIdx.x) + threadIdx.x;
            if (idx < size)
            {
                r[idx] = x[idx] - y[idx];
            }
        }
#endif
    }
}

extern "C" void cuda_difference_two_float(void * x, void * y, unsigned long size, unsigned long blocksize)
{
    dim3 grid;
    dim3 block;
    block.x = blocksize;
    grid.x = (unsigned)ceil(sqrt(size/(double)block.x));
    grid.y = grid.x;
    float * x_gpu((float *)x);
    float * y_gpu((float *)y);

    honei::cuda::difference_gpu<<<grid, block>>>(x_gpu, y_gpu, size);

    CUDA_ERROR();
}

#ifdef HONEI_CUDA_DOUBLE
extern "C" void cuda_difference_two_double(void * x, void * y, unsigned long size, unsigned long blocksize)
{
    dim3 grid;
    dim3 block;
    block.x = blocksize;
    grid.x = (unsigned)ceil(sqrt(size/(double)block.x));
    grid.y = grid.x;
    double * x_gpu((double *)x);
    double * y_gpu((double *)y);

    honei::cuda::difference_gpu<<<grid, block>>>(x_gpu, y_gpu, size);

    CUDA_ERROR();
}
#endif

extern "C" void cuda_difference_three_float(void * r, const void * x, void * y, unsigned long size, unsigned long blocksize)
{
    dim3 grid;
    dim3 block;
    block.x = blocksize;
    grid.x = (unsigned)ceil(sqrt(size/(double)block.x));
    grid.y = grid.x;
    float * r_gpu((float *)r);
    float * x_gpu((float *)x);
    float * y_gpu((float *)y);

    honei::cuda::difference_gpu<<<grid, block>>>(r_gpu, x_gpu, y_gpu, size);

    CUDA_ERROR();
}

#ifdef HONEI_CUDA_DOUBLE
extern "C" void cuda_difference_three_double(void * r, const void * x, void * y, unsigned long size, unsigned long blocksize)
{
    dim3 grid;
    dim3 block;
    block.x = blocksize;
    grid.x = (unsigned)ceil(sqrt(size/(double)block.x));
    grid.y = grid.x;
    double * r_gpu((double *)r);
    double * x_gpu((double *)x);
    double * y_gpu((double *)y);

    honei::cuda::difference_gpu<<<grid, block>>>(r_gpu, x_gpu, y_gpu, size);

    CUDA_ERROR();
}
#endif
