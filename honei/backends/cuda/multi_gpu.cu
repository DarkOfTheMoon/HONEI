/* vim: set sw=4 sts=4 et foldmethod=syntax : */

/*
 * Copyright (c) 2010 Dirk Ribbrock <dirk.ribbrock@uni-dortmund.de>
 *
 * This file is part of the HONEI C++ library. HONEI is free software;
 * you can redistribute it and/or modify it under the terms of the GNU General
 * Public License version 2, as published by the Free Software Foundation.
 *
 * HONEI is distributed in the hope that it will be useful, but WITHOUT ANY
 * WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more
 * details.
 *
 * You should have received a copy of the GNU General Public License along with
 * this program; if not, write to the Free Software Foundation, Inc., 59 Temple
 * Place, Suite 330, Boston, MA  02111-1307  USA
 */

#include <honei/backends/cuda/cuda_util.hh>
#include <stdio.h>

extern "C" int cuda_device_count()
{
    int device_count;
    hipGetDeviceCount(&device_count);
    CUDA_ERROR();
    return device_count;
}

extern "C" void cuda_print_device_name(int device)
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    printf("%s", prop.name);
}

extern "C" int cuda_get_device()
{
    int device(4711);
    hipGetDevice(&device);
    return device;
}

extern "C" void cuda_set_device(int device)
{
    hipSetDevice(device);
}
