#include "hip/hip_runtime.h"
/* vim: set sw=4 sts=4 et nofoldenable : */

/*
 * Copyright (c) 2008 Dirk Ribbrock <dirk.ribbrock@uni-dortmund.de>
 *
 * This file is part of the HONEI C++ library. HONEI is free software;
 * you can redistribute it and/or modify it under the terms of the GNU General
 * Public License version 2, as published by the Free Software Foundation.
 *
 * HONEI is distributed in the hope that it will be useful, but WITHOUT ANY
 * WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more
 * details.
 *
 * You should have received a copy of the GNU General Public License along with
 * this program; if not, write to the Free Software Foundation, Inc., 59 Temple
 * Place, Suite 330, Boston, MA  02111-1307  USA
 */

#include <honei/backends/cuda/cuda_util.hh>

namespace honei
{
    namespace cuda
    {
        __global__ void up_vel_dir_grid_gpu(
                float ** fs,
                unsigned long * types,
                unsigned long size)
        {
            unsigned long idx = (blockDim.y * blockIdx.y * gridDim.x * blockDim.x) + (blockDim.x * blockIdx.x) + threadIdx.x;
            if (idx < size)
            {
                unsigned long i(idx);
                if((types[i] & 1<<0) == 1<<0)
                    fs[4][i] = fs[0][i];
                if((types[i] & 1<<1) == 1<<1)
                    fs[5][i] = fs[1][i];
                if((types[i] & 1<<2) == 1<<2)
                    fs[6][i] = fs[2][i];
                if((types[i] & 1<<3) == 1<<3)
                    fs[7][i] = fs[3][i];
                if((types[i] & 1<<4) == 1<<4)
                    fs[0][i] = fs[4][i];
                if((types[i] & 1<<5) == 1<<5)
                    fs[1][i] = fs[5][i];
                if((types[i] & 1<<6) == 1<<6)
                    fs[2][i] = fs[6][i];
                if((types[i] & 1<<7) == 1<<7)
                    fs[3][i] = fs[7][i];

                // Corners
                if((types[i] & 1<<2) == 1<<2 && (types[i] & 1<<4) == 1<<4)
                {
                    fs[1][i] = fs[7][i];
                    fs[5][i] = fs[7][i];
                }
                if((types[i] & 1<<4) == 1<<4 && (types[i] & 1<<6) == 1<<6)
                {
                    fs[3][i] = fs[1][i];
                    fs[7][i] = fs[1][i];
                }
                if((types[i] & 1<<0) == 1<<0 && (types[i] & 1<<6) == 1<<6)
                {
                    fs[1][i] = fs[3][i];
                    fs[5][i] = fs[3][i];
                }
                if((types[i] & 1<<0) == 1<<0 && (types[i] & 1<<2) == 1<<2)
                {
                    fs[3][i] = fs[5][i];
                    fs[7][i] = fs[5][i];
                }
            }
        }
    }
}

extern "C" void cuda_up_vel_dir_grid_float(void * types,
        void * f_temp_1, void * f_temp_2,
        void * f_temp_3, void * f_temp_4, void * f_temp_5,
        void * f_temp_6, void * f_temp_7, void * f_temp_8,
        unsigned long size,
        unsigned long blocksize)
{
    dim3 grid;
    dim3 block;
    block.x = blocksize;
    grid.x = (unsigned)ceil(sqrt(size/(double)block.x));
    grid.y = grid.x;


    unsigned long * types_gpu((unsigned long *)types);

    float * f_temp_1_gpu((float *)f_temp_1);
    float * f_temp_2_gpu((float *)f_temp_2);
    float * f_temp_3_gpu((float *)f_temp_3);
    float * f_temp_4_gpu((float *)f_temp_4);
    float * f_temp_5_gpu((float *)f_temp_5);
    float * f_temp_6_gpu((float *)f_temp_6);
    float * f_temp_7_gpu((float *)f_temp_7);
    float * f_temp_8_gpu((float *)f_temp_8);

    float * fs[8];
    fs[0] = f_temp_1_gpu;
    fs[1] = f_temp_2_gpu;
    fs[2] = f_temp_3_gpu;
    fs[3] = f_temp_4_gpu;
    fs[4] = f_temp_5_gpu;
    fs[5] = f_temp_6_gpu;
    fs[6] = f_temp_7_gpu;
    fs[7] = f_temp_8_gpu;

    float ** fs_gpu;
    hipMalloc((void **) &fs_gpu, sizeof(fs));
    hipMemcpy(fs_gpu, fs, sizeof(fs), hipMemcpyHostToDevice);

    honei::cuda::up_vel_dir_grid_gpu<<<grid, block>>>(
            fs_gpu,
            types_gpu,
            size);

    hipFree(fs_gpu);

    CUDA_ERROR();
}
