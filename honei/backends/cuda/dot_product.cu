#include "hip/hip_runtime.h"
/* vim: set sw=4 sts=4 et nofoldenable : */

/*
 * Copyright (c) 2008 Dirk Ribbrock <dirk.ribbrock@uni-dortmund.de>
 *
 * This file is part of the HONEI C++ library. HONEI is free software;
 * you can redistribute it and/or modify it under the terms of the GNU General
 * Public License version 2, as published by the Free Software Foundation.
 *
 * HONEI is distributed in the hope that it will be useful, but WITHOUT ANY
 * WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more
 * details.
 *
 * You should have received a copy of the GNU General Public License along with
 * this program; if not, write to the Free Software Foundation, Inc., 59 Temple
 * Place, Suite 330, Boston, MA  02111-1307  USA
 */

#include <honei/backends/cuda/cuda_util.hh>

namespace honei
{
    namespace cuda
    {
        __global__ void dot_product_gpu(float * x, float * y, float * tmp, unsigned long size, unsigned long blocksize)
        {
            // calculate how many elements each thread needs to calculate
            const unsigned long iter = size / (blockDim.x * gridDim.x);
            int pos = blockIdx.x* blocksize + threadIdx.x;

            // clear the output
            tmp[blockIdx.x * blocksize + threadIdx.x] = 0;

            for (int i = 0 ; i < iter ; ++i)
            {
                tmp[blockIdx.x * blocksize + threadIdx.x] += x[pos] * y[pos];
                pos += blockDim.x * gridDim.x;
            }

            // for the last iteration, check if the elements are still available
            if (pos < size)
            {
                tmp[blockIdx.x * blocksize + threadIdx.x] += x[pos] * y[pos];
            }
        }
    }
}

extern "C" float cuda_dot_product_two_float(const float * x, const float * y, unsigned long size, unsigned long blocksize,
        unsigned long gridsize)
{
    float result(0.);

    if (size < gridsize * blocksize)
    {
        for (unsigned long i(0) ; i < size ; ++i)
        {
            result += x[i] * y[i];
        }
    }
    else
    {
        dim3 grid(gridsize);
        dim3 block(blocksize);
        float * x_gpu(0);
        float * y_gpu(0);
        float * tmp_cpu(0);
        float * tmp_gpu(0);

        hipMalloc((void**)&x_gpu, size * sizeof(float));
        hipMalloc((void**)&y_gpu, size * sizeof(float));
        hipMalloc((void**)&tmp_gpu, gridsize * blocksize * sizeof(float));
        hipHostMalloc((void**)&tmp_cpu, gridsize * blocksize * sizeof(float));

        hipMemcpy(x_gpu, x, size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(y_gpu, y, size * sizeof(float), hipMemcpyHostToDevice);

        honei::cuda::dot_product_gpu<<<grid, block>>>(x_gpu, y_gpu, tmp_gpu, size, blocksize);

        hipMemcpy(tmp_cpu, tmp_gpu, blocksize * gridsize * sizeof(float), hipMemcpyDeviceToHost);
        for (unsigned long i(0) ; i < blocksize * gridsize ; ++i)
        {
            result += tmp_cpu[i];
        }

        hipFree(x_gpu);
        hipFree(y_gpu);
        hipFree(tmp_gpu);
        hipHostFree(tmp_cpu);

        CUDA_ERROR();
    }
    return result;
}
