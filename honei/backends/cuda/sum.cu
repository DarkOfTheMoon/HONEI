#include "hip/hip_runtime.h"
/* vim: set sw=4 sts=4 et nofoldenable : */

/*
 * Copyright (c) 2008 Dirk Ribbrock <dirk.ribbrock@uni-dortmund.de>
 *
 * This file is part of the HONEI C++ library. HONEI is free software;
 * you can redistribute it and/or modify it under the terms of the GNU General
 * Public License version 2, as published by the Free Software Foundation.
 *
 * HONEI is distributed in the hope that it will be useful, but WITHOUT ANY
 * WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more
 * details.
 *
 * You should have received a copy of the GNU General Public License along with
 * this program; if not, write to the Free Software Foundation, Inc., 59 Temple
 * Place, Suite 330, Boston, MA  02111-1307  USA
 */

#include <honei/backends/cuda/cuda_util.hh>

namespace honei
{
    namespace cuda
    {
        __global__ void sum_gpu(float * x, float * y, unsigned long size)
        {
            int idx = blockDim.x *blockIdx.x + threadIdx.x;
            x[idx] = x[idx] + y[idx];
        }
    }
}

extern "C" void cuda_sum_two_float(float * x, float * y, unsigned long size, unsigned long blocksize)
{
    dim3 grid;
    dim3 block;
    block.x = blocksize;
    grid.x = ceil(size/(float)block.x);
    float * x_gpu(0);
    float * y_gpu(0);

    hipMalloc((void**)&x_gpu, size * sizeof(float));
    hipMalloc((void**)&y_gpu, size * sizeof(float));

    hipMemcpy(x_gpu, x, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y_gpu, y, size * sizeof(float), hipMemcpyHostToDevice);

    honei::cuda::sum_gpu<<<grid, block>>>(x_gpu, y_gpu, size);

    hipMemcpy(x, x_gpu, size * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(x_gpu);
    hipFree(y_gpu);

    CUDA_ERROR();
}
