#include "hip/hip_runtime.h"
/* vim: set sw=4 sts=4 et nofoldenable : */

/*
 * Copyright (c) 2008 Dirk Ribbrock <dirk.ribbrock@uni-dortmund.de>
 *
 * This file is part of the HONEI C++ library. HONEI is free software;
 * you can redistribute it and/or modify it under the terms of the GNU General
 * Public License version 2, as published by the Free Software Foundation.
 *
 * HONEI is distributed in the hope that it will be useful, but WITHOUT ANY
 * WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more
 * details.
 *
 * You should have received a copy of the GNU General Public License along with
 * this program; if not, write to the Free Software Foundation, Inc., 59 Temple
 * Place, Suite 330, Boston, MA  02111-1307  USA
 */

#include <limits>
#include <honei/backends/cuda/cuda_util.hh>

namespace honei
{
    namespace cuda
    {
        __global__ void force_grid_gpu_x(
                unsigned long * dir_1,
                unsigned long * interdir_1,
                float * f_temp_1,
                float * h, float * b,
                float distribution_x,
                float g, float d_x, float d_y, float d_t,
                unsigned long size)
        {
            unsigned long idx = (blockDim.y * blockIdx.y * gridDim.x * blockDim.x) + (blockDim.x * blockIdx.x) + threadIdx.x;
            float force_multiplier(d_t / (6 * d_x * d_x / (d_t * d_t)));
            float gravity_multiplier(-g);
            float force_times_gravity(force_multiplier * gravity_multiplier);

            if (idx < size)
            {
                unsigned long i(idx);
                float x(0);
                if (interdir_1[i] < size)
                {
                    x = force_times_gravity * distribution_x * (h[i] + h[interdir_1[i]]) / float(2);
                }
                if (dir_1[i] < size)
                {
                    x *= (b[dir_1[i]] - b[i]) / d_x;
                }
                f_temp_1[i] += x;
            }
        }
        __global__ void force_grid_gpu_xy(
                unsigned long * dir_2,
                unsigned long * dir_1,
                unsigned long * dir_3,
                float * f_temp_2,
                float * h, float * b,
                float distribution_x, float distribution_y,
                float g, float d_x, float d_y, float d_t,
                unsigned long size)
        {
            unsigned long idx = (blockDim.y * blockIdx.y * gridDim.x * blockDim.x) + (blockDim.x * blockIdx.x) + threadIdx.x;
            float force_multiplier(d_t / (6 * d_x * d_x / (d_t * d_t)));
            float gravity_multiplier(-g);
            float force_times_gravity(force_multiplier * gravity_multiplier);

            if (idx < size)
            {
                unsigned long i(idx);
                float x(0);
                if (dir_2[i] < size)
                {
                    x = force_times_gravity * distribution_x * (h[i] + h[dir_2[i]]) / float(2);
                }
                if (dir_1[i] < size)
                {
                    x *= (b[dir_1[i]] - b[i]) / d_x;
                }
                f_temp_2[i] += x;
                float y(0);
                if (dir_2[i] < size)
                {
                    y = force_times_gravity * distribution_y * (h[i] + h[dir_2[i]]) / float(2);
                }
                if (dir_3[i] < size)
                {
                    y *= (b[dir_3[i]] - b[i]) / d_y;
                }
                f_temp_2[i] += y;
            }
        }
        __global__ void force_grid_gpu_y(
                unsigned long * dir_3,
                unsigned long * interdir_3,
                float * f_temp_3,
                float * h, float * b,
                float distribution_y,
                float g, float d_x, float d_y, float d_t,
                unsigned long size)
        {
            unsigned long idx = (blockDim.y * blockIdx.y * gridDim.x * blockDim.x) + (blockDim.x * blockIdx.x) + threadIdx.x;
            float force_multiplier(d_t / (6 * d_x * d_x / (d_t * d_t)));
            float gravity_multiplier(-g);
            float force_times_gravity(force_multiplier * gravity_multiplier);

            if (idx < size)
            {
                unsigned long i(idx);
                float y(0);
                if (interdir_3[i] < size)
                {
                    y = force_times_gravity * distribution_y * (h[i] + h[interdir_3[i]]) / float(2);
                }
                if (dir_3[i] < size)
                {
                    y *= (b[dir_3[i]] - b[i]) / d_y;
                }
                f_temp_3[i] += y;
            }
        }
    }
}

extern "C" void cuda_force_grid_float(
        void * dir_1, void * dir_2, void * dir_3, void * dir_4,
        void * dir_5, void * dir_6, void * dir_7, void * dir_8,
        void * h, void * b,
        void * f_temp_1, void * f_temp_2,
        void * f_temp_3, void * f_temp_4, void * f_temp_5,
        void * f_temp_6, void * f_temp_7, void * f_temp_8,
        float distribution_x_1, float distribution_y_1,
        float distribution_x_2, float distribution_y_2,
        float distribution_x_3, float distribution_y_3,
        float distribution_x_4, float distribution_y_4,
        float distribution_x_5, float distribution_y_5,
        float distribution_x_6, float distribution_y_6,
        float distribution_x_7, float distribution_y_7,
        float distribution_x_8, float distribution_y_8,
        float g, float d_x, float d_y, float d_t,
        unsigned long size,
        unsigned long blocksize)
{
    dim3 grid;
    dim3 block;
    block.x = blocksize;
    grid.x = (unsigned)ceil(sqrt(size/(double)block.x));
    grid.y = grid.x;

    unsigned long * dir_1_gpu((unsigned long *)dir_1);
    unsigned long * dir_2_gpu((unsigned long *)dir_2);
    unsigned long * dir_3_gpu((unsigned long *)dir_3);
    unsigned long * dir_4_gpu((unsigned long *)dir_4);
    unsigned long * dir_5_gpu((unsigned long *)dir_5);
    unsigned long * dir_6_gpu((unsigned long *)dir_6);
    unsigned long * dir_7_gpu((unsigned long *)dir_7);
    unsigned long * dir_8_gpu((unsigned long *)dir_8);

    float * h_gpu((float *)h);
    float * b_gpu((float *)b);

    float * f_temp_1_gpu((float *)f_temp_1);
    float * f_temp_2_gpu((float *)f_temp_2);
    float * f_temp_3_gpu((float *)f_temp_3);
    float * f_temp_4_gpu((float *)f_temp_4);
    float * f_temp_5_gpu((float *)f_temp_5);
    float * f_temp_6_gpu((float *)f_temp_6);
    float * f_temp_7_gpu((float *)f_temp_7);
    float * f_temp_8_gpu((float *)f_temp_8);

    honei::cuda::force_grid_gpu_x<<<grid, block>>>(
            dir_1_gpu, dir_1_gpu, f_temp_1_gpu,
            h_gpu, b_gpu,
            distribution_x_1,
            g, d_x, d_y, d_t,
            size);
    honei::cuda::force_grid_gpu_xy<<<grid, block>>>(
            dir_2_gpu, dir_1_gpu, dir_3_gpu, f_temp_2_gpu,
            h_gpu, b_gpu,
            distribution_x_2, distribution_y_2,
            g, d_x, d_y, d_t,
            size);
    honei::cuda::force_grid_gpu_y<<<grid, block>>>(
            dir_3_gpu, dir_3_gpu, f_temp_3_gpu,
            h_gpu, b_gpu,
            distribution_y_3,
            g, d_x, d_y, d_t,
            size);
    honei::cuda::force_grid_gpu_xy<<<grid, block>>>(
            dir_4_gpu, dir_1_gpu, dir_3_gpu, f_temp_4_gpu,
            h_gpu, b_gpu,
            distribution_x_4, distribution_y_4,
            g, d_x, d_y, d_t,
            size);
    honei::cuda::force_grid_gpu_x<<<grid, block>>>(
            dir_1_gpu, dir_5_gpu, f_temp_5_gpu,
            h_gpu, b_gpu,
            distribution_x_5,
            g, d_x, d_y, d_t,
            size);
    honei::cuda::force_grid_gpu_xy<<<grid, block>>>(
            dir_6_gpu, dir_1_gpu, dir_3_gpu, f_temp_6_gpu,
            h_gpu, b_gpu,
            distribution_x_6, distribution_y_6,
            g, d_x, d_y, d_t,
            size);
    honei::cuda::force_grid_gpu_y<<<grid, block>>>(
            dir_3_gpu, dir_7_gpu, f_temp_7_gpu,
            h_gpu, b_gpu,
            distribution_y_7,
            g, d_x, d_y, d_t,
            size);
    honei::cuda::force_grid_gpu_xy<<<grid, block>>>(
            dir_8_gpu, dir_1_gpu, dir_3_gpu, f_temp_8_gpu,
            h_gpu, b_gpu,
            distribution_x_8, distribution_y_8,
            g, d_x, d_y, d_t,
            size);

    CUDA_ERROR();
}

//-----------------------------------------------------------------------------------------------
namespace honei
{
    namespace cuda
    {
        __global__ void force_grid_gpu_x_2(
                float * f_temp,
                float * h, float * u, float * v,
                float distribution_x,
                float g, float d_x, float d_t, float manning, float epsilon,
                unsigned long size)
        {
            unsigned long idx = (blockDim.y * blockIdx.y * gridDim.x * blockDim.x) + (blockDim.x * blockIdx.x) + threadIdx.x;

            if (idx < size)
            {
                unsigned long i(idx);
                float force_multiplier(d_t / (float(6) * d_x * d_x / (d_t * d_t)));
                force_multiplier *= g;
                if ( (powf(h[i], float(1./3.))) > epsilon || (powf(h[i], float(1./3.))) < -epsilon )
                {
                    f_temp[i] -= force_multiplier * distribution_x * manning * manning *
                        u[i] * sqrtf(u[i] * u[i] + v[i] * v[i]) / (powf(h[i], float(1./3.)));
                }
            }
        }

        __global__ void force_grid_gpu_xy_2(
                float * f_temp,
                float * h, float * u, float * v,
                float distribution_x,
                float distribution_y,
                float g, float d_x, float d_t, float manning, float epsilon,
                unsigned long size)
        {
            unsigned long idx = (blockDim.y * blockIdx.y * gridDim.x * blockDim.x) + (blockDim.x * blockIdx.x) + threadIdx.x;

            if (idx < size)
            {
                unsigned long i(idx);
                float force_multiplier(d_t / (float(6) * d_x * d_x / (d_t * d_t)));
                force_multiplier *= g;
                if ( (powf(h[i], float(1./3.))) > epsilon || (powf(h[i], float(1./3.))) < -epsilon )
                {
                    f_temp[i] -= force_multiplier * distribution_x * manning * manning *
                        u[i] * sqrtf(u[i] * u[i] + v[i] * v[i]) / (powf(h[i], float(1./3.)));
                    f_temp[i] -= force_multiplier * distribution_y * manning * manning *
                        u[i] * sqrtf(u[i] * u[i] + v[i] * v[i]) / (powf(h[i], float(1./3.)));
                }
            }
        }

        extern "C" void cuda_force_grid_float_2(
                void * h, void * u, void * v,
                void * f_temp_1, void * f_temp_2,
                void * f_temp_3, void * f_temp_4, void * f_temp_5,
                void * f_temp_6, void * f_temp_7, void * f_temp_8,
                float distribution_x_1, float distribution_y_1,
                float distribution_x_2, float distribution_y_2,
                float distribution_x_3, float distribution_y_3,
                float distribution_x_4, float distribution_y_4,
                float distribution_x_5, float distribution_y_5,
                float distribution_x_6, float distribution_y_6,
                float distribution_x_7, float distribution_y_7,
                float distribution_x_8, float distribution_y_8,
                float g, float d_x, float d_y, float d_t, float manning,
                unsigned long size,
                unsigned long blocksize)
        {
            dim3 grid;
            dim3 block;
            block.x = blocksize;
            grid.x = (unsigned)ceil(sqrt(size/(double)block.x));
            grid.y = grid.x;

            float * h_gpu((float *)h);
            float * u_gpu((float *)u);
            float * v_gpu((float *)v);

            float * f_temp_1_gpu((float *)f_temp_1);
            float * f_temp_2_gpu((float *)f_temp_2);
            float * f_temp_3_gpu((float *)f_temp_3);
            float * f_temp_4_gpu((float *)f_temp_4);
            float * f_temp_5_gpu((float *)f_temp_5);
            float * f_temp_6_gpu((float *)f_temp_6);
            float * f_temp_7_gpu((float *)f_temp_7);
            float * f_temp_8_gpu((float *)f_temp_8);

            honei::cuda::force_grid_gpu_x_2<<<grid, block>>>(
                    f_temp_1_gpu,
                    h_gpu, u_gpu, v_gpu,
                    distribution_x_1,
                    g, d_x, d_t, manning, std::numeric_limits<float>::epsilon(),
                    size);
            honei::cuda::force_grid_gpu_xy_2<<<grid, block>>>(
                    f_temp_2_gpu,
                    h_gpu, u_gpu, v_gpu,
                    distribution_x_2,
                    distribution_y_2,
                    g, d_x, d_t, manning, std::numeric_limits<float>::epsilon(),
                    size);
            honei::cuda::force_grid_gpu_x_2<<<grid, block>>>(
                    f_temp_3_gpu,
                    h_gpu, u_gpu, v_gpu,
                    distribution_y_3,
                    g, d_x, d_t, manning, std::numeric_limits<float>::epsilon(),
                    size);
            honei::cuda::force_grid_gpu_xy_2<<<grid, block>>>(
                    f_temp_4_gpu,
                    h_gpu, u_gpu, v_gpu,
                    distribution_x_4,
                    distribution_y_4,
                    g, d_x, d_t, manning, std::numeric_limits<float>::epsilon(),
                    size);
            honei::cuda::force_grid_gpu_x_2<<<grid, block>>>(
                    f_temp_5_gpu,
                    h_gpu, u_gpu, v_gpu,
                    distribution_x_5,
                    g, d_x, d_t, manning, std::numeric_limits<float>::epsilon(),
                    size);
            honei::cuda::force_grid_gpu_xy_2<<<grid, block>>>(
                    f_temp_6_gpu,
                    h_gpu, u_gpu, v_gpu,
                    distribution_x_6,
                    distribution_y_6,
                    g, d_x, d_t, manning, std::numeric_limits<float>::epsilon(),
                    size);
            honei::cuda::force_grid_gpu_x_2<<<grid, block>>>(
                    f_temp_7_gpu,
                    h_gpu, u_gpu, v_gpu,
                    distribution_y_7,
                    g, d_x, d_t, manning, std::numeric_limits<float>::epsilon(),
                    size);
            honei::cuda::force_grid_gpu_xy_2<<<grid, block>>>(
                    f_temp_8_gpu,
                    h_gpu, u_gpu, v_gpu,
                    distribution_x_8,
                    distribution_y_8,
                    g, d_x, d_t, manning, std::numeric_limits<float>::epsilon(),
                    size);

            CUDA_ERROR();
        }
    }
}
