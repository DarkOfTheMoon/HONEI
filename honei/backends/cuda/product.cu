#include "hip/hip_runtime.h"
/* vim: set sw=4 sts=4 et nofoldenable : */

/*
 * Copyright (c) 2008 Dirk Ribbrock <dirk.ribbrock@uni-dortmund.de>
 *
 * This file is part of the HONEI C++ library. HONEI is free software;
 * you can redistribute it and/or modify it under the terms of the GNU General
 * Public License version 2, as published by the Free Software Foundation.
 *
 * HONEI is distributed in the hope that it will be useful, but WITHOUT ANY
 * WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more
 * details.
 *
 * You should have received a copy of the GNU General Public License along with
 * this program; if not, write to the Free Software Foundation, Inc., 59 Temple
 * Place, Suite 330, Boston, MA  02111-1307  USA
 */
#include <honei/backends/cuda/cuda_util.hh>

texture<float,1> tex_x_float_product;
texture<int2,1>  tex_x_double_product;

namespace honei
{
    namespace cuda
    {
        //__global__ void product_bmdv_q1_gpu(float * x, float * y, unsigned long size)
        // further optimised version: don't compute on zeros in offdiagonals
        __global__ void product_bmdv_q1_gpu(
                float* ll, float* ld, float* lu,
                float* dl, float * dd, float* du,
                float* ul, float* ud, float* uu,
                float * x, float * y, unsigned long n, unsigned long m)
        {
            extern __shared__ float  smvf_cache[];

            unsigned long idx = blockDim.x*blockIdx.x+threadIdx.x;

            // runs from 0 to blockDim.x-1
            unsigned long lindex = threadIdx.x;

            float* Dcache = smvf_cache;
            float* Lcache = smvf_cache + blockDim.x + 2;
            float* Ucache = smvf_cache + 2 * (blockDim.x + 2);

            // prefetch chunks from iteration vector
            //
            //
            // data needed for DD, DU, DL: each thread loads one element, the first and last one load the border cases
            // x_0 ... x_blockdim-1 into c_1...c_blockdim
            if (idx < n) Dcache[lindex + 1] = x[idx];
            if (idx >= m && idx - m < n) Lcache[lindex + 1] = x[idx - m];
            if (idx + m < n) Ucache[lindex + 1] = x[idx + m];
            if (lindex == 0)
            {
                // x_-1 in c_0
                if (blockDim.x * blockIdx.x - 1 < n) Dcache[0] = x[blockDim.x * blockIdx.x - 1];
                if (blockDim.x * blockIdx.x - m - 1 < n) Lcache[0] = x[blockDim.x * blockIdx.x - m - 1];
                if (blockDim.x * blockIdx.x + m - 1 < n) Ucache[0] = x[blockDim.x * blockIdx.x + m - 1];
            }
            if (lindex == blockDim.x - 1)
            {
                // x_blockdim in c_blockdim+1
                if (blockDim.x * (blockIdx.x + 1) < n) Dcache[blockDim.x + 1] = x[blockDim.x * (blockIdx.x + 1)];
                if (blockDim.x * (blockIdx.x + 1) - m < n) Lcache[blockDim.x + 1] = x[blockDim.x * (blockIdx.x + 1) - m];
                if (blockDim.x * (blockIdx.x + 1) + m  < n) Ucache[blockDim.x + 1] = x[blockDim.x * (blockIdx.x + 1) + m];
            }
            __syncthreads();
            // now, compute
            if (idx < n)
            {
                float ytemp1 = dd[idx] * Dcache[lindex + 1];
                if (idx > 0) ytemp1 += dl[idx] * Dcache[lindex];
                if (idx < n - 1) ytemp1 += du[idx] * Dcache[lindex + 2];

                if (idx > m) ytemp1 += ll[idx] * Lcache[lindex];
                if (idx > m - 1) ytemp1 += ld[idx] * Lcache[lindex + 1];
                if (idx > m - 2) ytemp1 += lu[idx] * Lcache[lindex + 2];

                if (idx < n - m + 1) ytemp1 += ul[idx] * Ucache[lindex];
                if (idx < n - m) ytemp1 += ud[idx] * Ucache[lindex + 1];
                if (idx < n - m - 1) ytemp1 += uu[idx] * Ucache[lindex + 2];
                y[idx] = ytemp1;
            }
        }

        #ifdef HONEI_CUDA_DOUBLE
        __global__ void product_bmdv_q1_gpu(
                double* ll, double* ld, double* lu,
                double* dl, double * dd, double* du,
                double* ul, double* ud, double* uu,
                double * x, double * y, unsigned long n, unsigned long m)
        {
            extern __shared__ double  smvd_cache[];

            unsigned long idx = blockDim.x*blockIdx.x+threadIdx.x;

            // runs from 0 to blockDim.x-1
            unsigned long lindex = threadIdx.x;

            double* Dcache = smvd_cache;
            double* Lcache = smvd_cache + blockDim.x + 2;
            double* Ucache = smvd_cache + 2 * (blockDim.x + 2);

            // prefetch chunks from iteration vector
            //
            //
            // data needed for DD, DU, DL: each thread loads one element, the first and last one load the border cases
            // x_0 ... x_blockdim-1 into c_1...c_blockdim
            if (idx < n) Dcache[lindex + 1] = x[idx];
            if (idx >= m && idx - m < n) Lcache[lindex + 1] = x[idx - m];
            if (idx + m < n) Ucache[lindex + 1] = x[idx + m];
            if (lindex == 0)
            {
                // x_-1 in c_0
                if (blockDim.x * blockIdx.x - 1 < n) Dcache[0] = x[blockDim.x * blockIdx.x - 1];
                if (blockDim.x * blockIdx.x - m - 1 < n) Lcache[0] = x[blockDim.x * blockIdx.x - m - 1];
                if (blockDim.x * blockIdx.x + m - 1 < n) Ucache[0] = x[blockDim.x * blockIdx.x + m - 1];
            }
            if (lindex == blockDim.x - 1)
            {
                // x_blockdim in c_blockdim+1
                if (blockDim.x * (blockIdx.x + 1) < n) Dcache[blockDim.x + 1] = x[blockDim.x * (blockIdx.x + 1)];
                if (blockDim.x * (blockIdx.x + 1) - m < n) Lcache[blockDim.x + 1] = x[blockDim.x * (blockIdx.x + 1) - m];
                if (blockDim.x * (blockIdx.x + 1) + m  < n) Ucache[blockDim.x + 1] = x[blockDim.x * (blockIdx.x + 1) + m];
            }
            __syncthreads();
            // now, compute
            if (idx < n)
            {
                double ytemp1 = dd[idx] * Dcache[lindex + 1];
                if (idx > 0) ytemp1 += dl[idx] * Dcache[lindex];
                if (idx < n - 1) ytemp1 += du[idx] * Dcache[lindex + 2];

                if (idx > m) ytemp1 += ll[idx] * Lcache[lindex];
                if (idx > m - 1) ytemp1 += ld[idx] * Lcache[lindex + 1];
                if (idx > m - 2) ytemp1 += lu[idx] * Lcache[lindex + 2];

                if (idx < n - m + 1) ytemp1 += ul[idx] * Ucache[lindex];
                if (idx < n - m) ytemp1 += ud[idx] * Ucache[lindex + 1];
                if (idx < n - m - 1) ytemp1 += uu[idx] * Ucache[lindex + 2];
                y[idx] = ytemp1;
            }
        }
        #endif

        __global__ void product_smell_dv_gpu(float * x, float * y, const unsigned long * Aj, const float * Ax, const unsigned long * Arl,
                unsigned long row_start, unsigned long row_end, unsigned long num_cols_per_row, unsigned long stride, unsigned long threads)
        {
            extern __shared__ float  shared_ell_float[];

            const unsigned long T = threads;
            const unsigned long idx = blockDim.x*blockIdx.x+threadIdx.x + (row_start * T);
            const unsigned long idb = threadIdx.x;
            const unsigned long idp = idb % T;
            const unsigned long row = idx / T;

            if(row >= row_end){ return; }
            shared_ell_float[idb] = 0;
            float sum = float(0);

            const unsigned long max = Arl[row];
            Ax += (row*T)+idp;
            Aj += (row*T)+idp;
            for(unsigned long k = 0; k < max ; ++k)
            {
                //sum += value * x[col];
                sum += *Ax * tex1Dfetch(tex_x_float_product, *Aj);
                Ax += stride;
                Aj += stride;
            }
            shared_ell_float[idb] = sum;

            switch (threads)
            {
                case 32:
                    if (idp < 16)
                        shared_ell_float[idb] += shared_ell_float[idb + 16];
                case 16:
                    if (idp < 8)
                        shared_ell_float[idb] += shared_ell_float[idb + 8];
                case 8:
                    if (idp < 4)
                        shared_ell_float[idb] += shared_ell_float[idb + 4];
                case 4:
                    if (idp < 2)
                        shared_ell_float[idb] += shared_ell_float[idb + 2];
                case 2:
                    if (idp == 0)
                        y[row - row_start] = shared_ell_float[idb] + shared_ell_float[idb + 1];
                    break;
                case 1:
                    y[row - row_start] = shared_ell_float[idb];
                    break;
                default:
                    break;
            }
        }

#ifdef HONEI_CUDA_DOUBLE
        __global__ void product_smell_dv_gpu(double * x, double * y, const unsigned long * Aj, const double * Ax, const unsigned long * Arl,
                unsigned long row_start, unsigned long row_end, unsigned long num_cols_per_row, unsigned long stride, unsigned long threads)
        {
            extern __shared__ double  shared_ell_double[];

            const unsigned long T = threads;
            const unsigned long idx = blockDim.x*blockIdx.x+threadIdx.x + (row_start * T);
            const unsigned long idb = threadIdx.x;
            const unsigned long idp = idb % T;
            const unsigned long row = idx / T;

            if(row >= row_end){ return; }
            shared_ell_double[idb] = 0;
            double sum = double(0);

            const unsigned long max = Arl[row];
            Ax += (row*T)+idp;
            Aj += (row*T)+idp;
            for(unsigned long k = 0; k < max ; ++k)
            {
                //sum += value * x[col];
                int2 v = tex1Dfetch(tex_x_double_product, *Aj);
                sum += *Ax * __hiloint2double(v.y, v.x);
                Ax += stride;
                Aj += stride;
            }
            shared_ell_double[idb] = sum;

            switch (threads)
            {
                case 32:
                    if (idp < 16)
                        shared_ell_double[idb] += shared_ell_double[idb + 16];
                case 16:
                    if (idp < 8)
                        shared_ell_double[idb] += shared_ell_double[idb + 8];
                case 8:
                    if (idp < 4)
                        shared_ell_double[idb] += shared_ell_double[idb + 4];
                case 4:
                    if (idp < 2)
                        shared_ell_double[idb] += shared_ell_double[idb + 2];
                case 2:
                    if (idp == 0)
                        y[row - row_start] = shared_ell_double[idb] + shared_ell_double[idb + 1];
                    break;
                case 1:
                    y[row - row_start] = shared_ell_double[idb];
                    break;
                default:
                    break;
            }
        }
#endif

        __global__ void product_csr_dv_gpu(float * x, float * y, const unsigned long * Aj, const float * Ax, const unsigned long * Ar,
                unsigned long row_start, unsigned long row_end, unsigned long atomicsize)
        {
            unsigned long idx = blockDim.x*blockIdx.x+threadIdx.x;
            unsigned long row = idx + row_start;

            if(row >= row_end){ return; }

            float sum(0);
            const unsigned long end(Ar[row+1]);
            for (unsigned long i(Ar[row]) ; i < end ; ++i)
            {
                for (unsigned long blocki(0) ; blocki < atomicsize ; ++blocki)
                    sum += Ax[(i * atomicsize) + blocki] * tex1Dfetch(tex_x_float_product, Aj[i] + blocki);
            }
            y[row] = sum;
        }

#ifdef HONEI_CUDA_DOUBLE
        __global__ void product_csr_dv_gpu(double * x, double * y, const unsigned long * Aj, const double * Ax, const unsigned long * Ar,
                unsigned long row_start, unsigned long row_end, unsigned long atomicsize)
        {
            unsigned long idx = blockDim.x*blockIdx.x+threadIdx.x;
            unsigned long row = idx + row_start;

            if(row >= row_end){ return; }

            double sum(0);
            const unsigned long end(Ar[row+1]);
            for (unsigned long i(Ar[row]) ; i < end ; ++i)
            {
                for (unsigned long blocki(0) ; blocki < atomicsize ; ++blocki)
                {
                    int2 v = tex1Dfetch(tex_x_double_product, Aj[i] + blocki);
                    sum += Ax[(i * atomicsize) + blocki] * __hiloint2double(v.y, v.x);
                }
            }
            y[row] = sum;
        }
#endif
    }
}

extern "C" void cuda_product_bmdv_q1_float (void * ll, void * ld, void * lu,
        void * dl, void * dd, void *du,
        void * ul, void * ud, void *uu, void * x, void * y,
        unsigned long size, unsigned long blocksize, unsigned long m)
{
    dim3 grid;
    dim3 block;
    block.x = blocksize;
    grid.x = (unsigned)ceil(size/(double)(block.x));
    float * x_gpu((float *)x);
    float * y_gpu((float *)y);
    float * ll_gpu((float *)ll);
    float * ld_gpu((float *)ld);
    float * lu_gpu((float *)lu);
    float * dl_gpu((float *)dl);
    float * dd_gpu((float *)dd);
    float * du_gpu((float *)du);
    float * ul_gpu((float *)ul);
    float * ud_gpu((float *)ud);
    float * uu_gpu((float *)uu);


    honei::cuda::product_bmdv_q1_gpu<<<grid, block, 3 * (block.x + 2 ) * sizeof(float)>>>(ll_gpu, ld_gpu, lu_gpu, dl_gpu, dd_gpu, du_gpu, ul_gpu, ud_gpu, uu_gpu, x_gpu, y_gpu, size, m);

    CUDA_ERROR();
}

#ifdef HONEI_CUDA_DOUBLE
extern "C" void cuda_product_bmdv_q1_double (void * ll, void * ld, void * lu,
        void * dl, void * dd, void *du,
        void * ul, void * ud, void *uu, void * x, void * y,
        unsigned long size, unsigned long blocksize, unsigned long m)
{
    dim3 grid;
    dim3 block;
    block.x = blocksize;
    grid.x = (unsigned)ceil(size/(double)(block.x));
    double * x_gpu((double *)x);
    double * y_gpu((double *)y);
    double * ll_gpu((double *)ll);
    double * ld_gpu((double *)ld);
    double * lu_gpu((double *)lu);
    double * dl_gpu((double *)dl);
    double * dd_gpu((double *)dd);
    double * du_gpu((double *)du);
    double * ul_gpu((double *)ul);
    double * ud_gpu((double *)ud);
    double * uu_gpu((double *)uu);


    honei::cuda::product_bmdv_q1_gpu<<<grid, block, 3 * (block.x + 2 ) * sizeof(double)>>>(ll_gpu, ld_gpu, lu_gpu, dl_gpu, dd_gpu, du_gpu, ul_gpu, ud_gpu, uu_gpu, x_gpu, y_gpu, size, m);

    CUDA_ERROR();
}
#endif

extern "C" void cuda_product_smell_dv_float(void * x, void * y, void * Aj, void * Ax, void * Arl,
        unsigned long row_start, unsigned long row_end, unsigned long num_cols_per_row,
        unsigned long stride, unsigned long blocksize, unsigned long threads, hipStream_t stream)
{
    dim3 grid;
    dim3 block;
    block.x = blocksize;
    grid.x = (unsigned)ceil((threads*(row_end - row_start))/(double)(block.x));

    float * x_gpu((float *)x);
    float * y_gpu((float *)y);
    unsigned long * Aj_gpu((unsigned long *)Aj);
    float * Ax_gpu((float *)Ax);
    unsigned long * Arl_gpu((unsigned long *)Arl);

    hipBindTexture(NULL, tex_x_float_product, x_gpu);
    honei::cuda::product_smell_dv_gpu<<<grid, block, block.x * sizeof(float), stream>>>(x_gpu, y_gpu, Aj_gpu, Ax_gpu, Arl_gpu,
            row_start, row_end, num_cols_per_row, stride, threads);
    hipUnbindTexture(tex_x_float_product);

    CUDA_ERROR();
}

#ifdef HONEI_CUDA_DOUBLE
extern "C" void cuda_product_smell_dv_double(void * x, void * y, void * Aj, void * Ax, void * Arl,
        unsigned long row_start, unsigned long row_end, unsigned long num_cols_per_row,
        unsigned long stride, unsigned long blocksize, unsigned long threads, hipStream_t stream)
{
    dim3 grid;
    dim3 block;
    block.x = blocksize;
    grid.x = (unsigned)ceil((threads*(row_end - row_start))/(double)(block.x));

    double * x_gpu((double *)x);
    double * y_gpu((double *)y);
    unsigned long * Aj_gpu((unsigned long *)Aj);
    double * Ax_gpu((double *)Ax);
    unsigned long * Arl_gpu((unsigned long *)Arl);

    hipBindTexture(NULL, tex_x_double_product, x_gpu);
    honei::cuda::product_smell_dv_gpu<<<grid, block, block.x * sizeof(double), stream>>>(x_gpu, y_gpu, Aj_gpu, Ax_gpu, Arl_gpu,
            row_start, row_end, num_cols_per_row, stride, threads);
    hipUnbindTexture(tex_x_double_product);

    CUDA_ERROR();
}
#endif

extern "C" void cuda_product_csr_dv_float(void * x, void * y, void * Aj, void * Ax, void * Ar,
        unsigned long row_start, unsigned long row_end, unsigned long atomicsize, unsigned long blocksize, hipStream_t stream)
{
    dim3 grid;
    dim3 block;
    block.x = blocksize;
    grid.x = (unsigned)ceil((row_end - row_start)/(double)(block.x));

    float * x_gpu((float *)x);
    float * y_gpu((float *)y);
    unsigned long * Aj_gpu((unsigned long *)Aj);
    float * Ax_gpu((float *)Ax);
    unsigned long * Ar_gpu((unsigned long *)Ar);

    hipBindTexture(NULL, tex_x_float_product, x_gpu);
    honei::cuda::product_csr_dv_gpu<<<grid, block, block.x * sizeof(float), stream>>>(x_gpu, y_gpu, Aj_gpu, Ax_gpu, Ar_gpu,
            row_start, row_end, atomicsize);
    hipUnbindTexture(tex_x_float_product);

    CUDA_ERROR();
}

#ifdef HONEI_CUDA_DOUBLE
extern "C" void cuda_product_csr_dv_double(void * x, void * y, void * Aj, void * Ax, void * Ar,
        unsigned long row_start, unsigned long row_end, unsigned long atomicsize, unsigned long blocksize, hipStream_t stream)
{
    dim3 grid;
    dim3 block;
    block.x = blocksize;
    grid.x = (unsigned)ceil((row_end - row_start)/(double)(block.x));

    double * x_gpu((double *)x);
    double * y_gpu((double *)y);
    unsigned long * Aj_gpu((unsigned long *)Aj);
    double * Ax_gpu((double *)Ax);
    unsigned long * Ar_gpu((unsigned long *)Ar);

    hipBindTexture(NULL, tex_x_double_product, x_gpu);
    honei::cuda::product_csr_dv_gpu<<<grid, block, block.x * sizeof(double), stream>>>(x_gpu, y_gpu, Aj_gpu, Ax_gpu, Ar_gpu,
            row_start, row_end, atomicsize);
    hipUnbindTexture(tex_x_double_product);

    CUDA_ERROR();
}
#endif
