#include "hip/hip_runtime.h"
/* vim: set sw=4 sts=4 et nofoldenable : */

/*
 * Copyright (c) 2008 Dirk Ribbrock <dirk.ribbrock@uni-dortmund.de>
 *
 * This file is part of the HONEI C++ library. HONEI is free software;
 * you can redistribute it and/or modify it under the terms of the GNU General
 * Public License version 2, as published by the Free Software Foundation.
 *
 * HONEI is distributed in the hope that it will be useful, but WITHOUT ANY
 * WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more
 * details.
 *
 * You should have received a copy of the GNU General Public License along with
 * this program; if not, write to the Free Software Foundation, Inc., 59 Temple
 * Place, Suite 330, Boston, MA  02111-1307  USA
 */

#include <honei/backends/cuda/cuda_util.hh>

namespace honei
{
    namespace cuda
    {
        __global__ void scale_gpu(float * x, float a, unsigned long size)
        {
            int idx = (blockDim.y * blockIdx.y * gridDim.x * blockDim.x) + (blockDim.x * blockIdx.x) + threadIdx.x;
            if (idx < size)
            {
                x[idx] = x[idx] * a;
            }
        }
    }
}

extern "C" void cuda_scale_one_float(void * x, float a, unsigned long size, unsigned long blocksize)
{
    dim3 grid;
    dim3 block;
    block.x = blocksize;
    grid.x = ceil(sqrt(size/(double)block.x));
    grid.y = grid.x;
    float * x_gpu((float *)x);

    honei::cuda::scale_gpu<<<grid, block>>>(x_gpu, a, size);

    CUDA_ERROR();
}
