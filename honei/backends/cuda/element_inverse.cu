#include "hip/hip_runtime.h"
/* vim: set sw=4 sts=4 et nofoldenable : */

/*
 * Copyright (c) 2008 Dirk Ribbrock <dirk.ribbrock@uni-dortmund.de>
 *
 * This file is part of the HONEI C++ library. HONEI is free software;
 * you can redistribute it and/or modify it under the terms of the GNU General
 * Public License version 2, as published by the Free Software Foundation.
 *
 * HONEI is distributed in the hope that it will be useful, but WITHOUT ANY
 * WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more
 * details.
 *
 * You should have received a copy of the GNU General Public License along with
 * this program; if not, write to the Free Software Foundation, Inc., 59 Temple
 * Place, Suite 330, Boston, MA  02111-1307  USA
 */

#include <honei/backends/cuda/cuda_util.hh>

namespace honei
{
    namespace cuda
    {
        __global__ void element_inverse_gpu(float * x, unsigned long size)
        {
            int idx = blockDim.x *blockIdx.x + threadIdx.x;
            if (x[idx] != 0)
                x[idx] = 1 / x[idx];
        }
    }
}

extern "C" void cuda_element_inverse_one_float(float * x, unsigned long size, unsigned long blocksize)
{
    dim3 grid;
    dim3 block;
    block.x = blocksize;
    grid.x = ceil(size/(float)block.x);
    float * x_gpu(0);

    hipMalloc((void**)&x_gpu, size * sizeof(float));

    hipMemcpy(x_gpu, x, size * sizeof(float), hipMemcpyHostToDevice);

    honei::cuda::element_inverse_gpu<<<grid, block, block.x * sizeof(float)>>>(x_gpu, size);

    hipMemcpy(x, x_gpu, size * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(x_gpu);

    CUDA_ERROR();
}
