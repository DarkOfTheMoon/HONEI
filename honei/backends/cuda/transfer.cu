
#include <hip/hip_runtime.h>
/* vim: set sw=4 sts=4 et foldmethod=syntax : */

/*
 * Copyright (c) 2008 Dirk Ribbrock <dirk.ribbrock@uni-dortmund.de>
 *
 * This file is part of the HONEI C++ library. HONEI is free software;
 * you can redistribute it and/or modify it under the terms of the GNU General
 * Public License version 2, as published by the Free Software Foundation.
 *
 * HONEI is distributed in the hope that it will be useful, but WITHOUT ANY
 * WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more
 * details.
 *
 * You should have received a copy of the GNU General Public License along with
 * this program; if not, write to the Free Software Foundation, Inc., 59 Temple
 * Place, Suite 330, Boston, MA  02111-1307  USA
 */

extern "C" unsigned long cuda_upload(unsigned long src, unsigned long bytes)
{
    unsigned long gpu;
    hipMalloc((void**)&gpu, bytes);
    hipMemcpy((void*)&gpu, (void*)&src, bytes, hipMemcpyHostToDevice);
    return gpu;
}

extern "C" void cuda_download(unsigned long src, unsigned long target, unsigned long bytes)
{
    hipMemcpy((void*)&target, (void*)&src, bytes, hipMemcpyDeviceToHost);
}

extern "C" void cuda_free(unsigned long src)
{
    hipFree((void*)&src);
}

